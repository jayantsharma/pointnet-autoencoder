#define EIGEN_DEFAULT_DENSE_INDEX_TYPE int32_t

// #include <bits/stdc++.h>

#include <hip/hip_runtime.h>
#include <iostream>
#include <fstream>
#include <sstream>
#include <vector>
#include <string>
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <algorithm>
#include <list>
#include <unistd.h>

using namespace std;

#define SIGN(a, b) ((b) >= 0.0 ? fabs(a) : -fabs(a))
#define MAX(x,y) ((x)>(y)?(x):(y))

const int NUM_NBRS = 10;

///////////////////////////////////////////////////////

__host__ __device__ static double PYTHAG(double a, double b)
{
    double at = fabs(a), bt = fabs(b), ct, result;

    if (at > bt)       { ct = bt / at; result = at * sqrt(1.0 + ct * ct); }
    else if (bt > 0.0) { ct = at / bt; result = bt * sqrt(1.0 + ct * ct); }
    else result = 0.0;
    return(result);
}

///////////////////////////////////////////////////////

__host__ __device__
int dsvd(float a[][NUM_NBRS], int m, int n, float w[3], float v[][3]){
    int flag, i, its, j, jj, k, l, nm;
    double c, f, h, s, x, y, z;
    double anorm = 0.0, g = 0.0, scale = 0.0;
    double *rv1;
  
    if (m < n) 
    {
        // fprintf(stderr, "#rows must be > #cols \n");
        return(0);
    }
  
    rv1 = (double *)malloc((unsigned int) n*sizeof(double));

/* Householder reduction to bidiagonal form */
    for (i = 0; i < n; i++) 
    {
        /* left-hand reduction */
        l = i + 1;
        rv1[i] = scale * g;
        g = s = scale = 0.0;
        if (i < m) 
        {
            for (k = i; k < m; k++) 
                scale += fabs((double)a[k][i]);
            if (scale) 
            {
                for (k = i; k < m; k++) 
                {
                    a[k][i] = (float)((double)a[k][i]/scale);
                    s += ((double)a[k][i] * (double)a[k][i]);
                }
                f = (double)a[i][i];
                g = -SIGN(sqrt(s), f);
                h = f * g - s;
                a[i][i] = (float)(f - g);
                if (i != n - 1) 
                {
                    for (j = l; j < n; j++) 
                    {
                        for (s = 0.0, k = i; k < m; k++) 
                            s += ((double)a[k][i] * (double)a[k][j]);
                        f = s / h;
                        for (k = i; k < m; k++) 
                            a[k][j] += (float)(f * (double)a[k][i]);
                    }
                }
                for (k = i; k < m; k++) 
                    a[k][i] = (float)((double)a[k][i]*scale);
            }
        }
        w[i] = (float)(scale * g);
    
        /* right-hand reduction */
        g = s = scale = 0.0;
        if (i < m && i != n - 1) 
        {
            for (k = l; k < n; k++) 
                scale += fabs((double)a[i][k]);
            if (scale) 
            {
                for (k = l; k < n; k++) 
                {
                    a[i][k] = (float)((double)a[i][k]/scale);
                    s += ((double)a[i][k] * (double)a[i][k]);
                }
                f = (double)a[i][l];
                g = -SIGN(sqrt(s), f);
                h = f * g - s;
                a[i][l] = (float)(f - g);
                for (k = l; k < n; k++) 
                    rv1[k] = (double)a[i][k] / h;
                if (i != m - 1) 
                {
                    for (j = l; j < m; j++) 
                    {
                        for (s = 0.0, k = l; k < n; k++) 
                            s += ((double)a[j][k] * (double)a[i][k]);
                        for (k = l; k < n; k++) 
                            a[j][k] += (float)(s * rv1[k]);
                    }
                }
                for (k = l; k < n; k++) 
                    a[i][k] = (float)((double)a[i][k]*scale);
            }
        }
        anorm = MAX(anorm, (fabs((double)w[i]) + fabs(rv1[i])));
    }
  
    /* accumulate the right-hand transformation */
    for (i = n - 1; i >= 0; i--) 
    {
        if (i < n - 1) 
        {
            if (g) 
            {
                for (j = l; j < n; j++)
                    v[j][i] = (float)(((double)a[i][j] / (double)a[i][l]) / g);
                    /* double division to avoid underflow */
                for (j = l; j < n; j++) 
                {
                    for (s = 0.0, k = l; k < n; k++) 
                        s += ((double)a[i][k] * (double)v[k][j]);
                    for (k = l; k < n; k++) 
                        v[k][j] += (float)(s * (double)v[k][i]);
                }
            }
            for (j = l; j < n; j++) 
                v[i][j] = v[j][i] = 0.0;
        }
        v[i][i] = 1.0;
        g = rv1[i];
        l = i;
    }
  
    /* accumulate the left-hand transformation */
    for (i = n - 1; i >= 0; i--) 
    {
        l = i + 1;
        g = (double)w[i];
        if (i < n - 1) 
            for (j = l; j < n; j++) 
                a[i][j] = 0.0;
        if (g) 
        {
            g = 1.0 / g;
            if (i != n - 1) 
            {
                for (j = l; j < n; j++) 
                {
                    for (s = 0.0, k = l; k < m; k++) 
                        s += ((double)a[k][i] * (double)a[k][j]);
                    f = (s / (double)a[i][i]) * g;
                    for (k = i; k < m; k++) 
                        a[k][j] += (float)(f * (double)a[k][i]);
                }
            }
            for (j = i; j < m; j++) 
                a[j][i] = (float)((double)a[j][i]*g);
        }
        else 
        {
            for (j = i; j < m; j++) 
                a[j][i] = 0.0;
        }
        ++a[i][i];
    }

    /* diagonalize the bidiagonal form */
    for (k = n - 1; k >= 0; k--) 
    {                             /* loop over singular values */
        for (its = 0; its < 30; its++) 
        {                         /* loop over allowed iterations */
            flag = 1;
            for (l = k; l >= 0; l--) 
            {                     /* test for splitting */
                nm = l - 1;
                if (fabs(rv1[l]) + anorm == anorm) 
                {
                    flag = 0;
                    break;
                }
                if (fabs((double)w[nm]) + anorm == anorm) 
                    break;
            }
            if (flag) 
            {
                c = 0.0;
                s = 1.0;
                for (i = l; i <= k; i++) 
                {
                    f = s * rv1[i];
                    if (fabs(f) + anorm != anorm) 
                    {
                        g = (double)w[i];
                        h = PYTHAG(f, g);
                        w[i] = (float)h; 
                        h = 1.0 / h;
                        c = g * h;
                        s = (- f * h);
                        for (j = 0; j < m; j++) 
                        {
                            y = (double)a[j][nm];
                            z = (double)a[j][i];
                            a[j][nm] = (float)(y * c + z * s);
                            a[j][i] = (float)(z * c - y * s);
                        }
                    }
                }
            }
            z = (double)w[k];
            if (l == k) 
            {                  /* convergence */
                if (z < 0.0) 
                {              /* make singular value nonnegative */
                    w[k] = (float)(-z);
                    for (j = 0; j < n; j++) 
                        v[j][k] = (-v[j][k]);
                }
                break;
            }
            if (its >= 30) {
                free((void*) rv1);
                // fprintf(stderr, "No convergence after 30,000! iterations \n");
                return(0);
            }
    
            /* shift from bottom 2 x 2 minor */
            x = (double)w[l];
            nm = k - 1;
            y = (double)w[nm];
            g = rv1[nm];
            h = rv1[k];
            f = ((y - z) * (y + z) + (g - h) * (g + h)) / (2.0 * h * y);
            g = PYTHAG(f, 1.0);
            f = ((x - z) * (x + z) + h * ((y / (f + SIGN(g, f))) - h)) / x;
          
            /* next QR transformation */
            c = s = 1.0;
            for (j = l; j <= nm; j++) 
            {
                i = j + 1;
                g = rv1[i];
                y = (double)w[i];
                h = s * g;
                g = c * g;
                z = PYTHAG(f, h);
                rv1[j] = z;
                c = f / z;
                s = h / z;
                f = x * c + g * s;
                g = g * c - x * s;
                h = y * s;
                y = y * c;
                for (jj = 0; jj < n; jj++) 
                {
                    x = (double)v[jj][j];
                    z = (double)v[jj][i];
                    v[jj][j] = (float)(x * c + z * s);
                    v[jj][i] = (float)(z * c - x * s);
                }
                z = PYTHAG(f, h);
                w[j] = (float)z;
                if (z) 
                {
                    z = 1.0 / z;
                    c = f * z;
                    s = h * z;
                }
                f = (c * g) + (s * y);
                x = (c * y) - (s * g);
                for (jj = 0; jj < m; jj++) 
                {
                    y = (double)a[jj][j];
                    z = (double)a[jj][i];
                    a[jj][j] = (float)(y * c + z * s);
                    a[jj][i] = (float)(z * c - y * s);
                }
            }
            rv1[l] = 0.0;
            rv1[k] = f;
            w[k] = (float)x;
        }
    }
    free((void*) rv1);
    return(1);
}

void read_clouds(float * result){
  // File pointer 
  fstream fin; 

  // Open an existing file 
  fin.open("clouds2.txt", ios::in); 

  // Read the Data from the file 
  // as String Vector 
  string line, word, temp; 

  int i=0;
  while (fin >> line) { 
    // used for breaking words 
    stringstream s(line); 

    while (getline(s, word, ',')) { 
      float num = stof(word);
      result[i] = num; 
      i++;
    } 
  } 
} 

static void nnsearch(int b,int n,int m,const float * xyz1,const float * xyz2,float * dist,int * idx){
    for (int i=0;i<b;i++){
        for (int j=0;j<n;j++){
            float x1=xyz1[(i*n+j)*3+0];
            float y1=xyz1[(i*n+j)*3+1];
            float z1=xyz1[(i*n+j)*3+2];
            double best=100;
            int besti=0;
            for (int k=0;k<m;k++){
                // if(k == j)  continue;
                float x2=xyz2[(i*m+k)*3+0]-x1;
                float y2=xyz2[(i*m+k)*3+1]-y1;
                float z2=xyz2[(i*m+k)*3+2]-z1;
                double d=x2*x2+y2*y2+z2*z2;
                if (d<best){
                    best=d;
                    besti=k;
                }
            }
            dist[i*n+j]=best;
            idx[i*n+j]=besti;
        }
    }
}

static void knearestnbr(int b,int n,const float * xyz,float * dist, int *idx){
    for (int i=0;i<b;i++){
      int num_nbrs = 10;
      for (int j=0;j<n;j++){
        float x1=xyz[(i*n+j)*3+0];
        float y1=xyz[(i*n+j)*3+1];
        float z1=xyz[(i*n+j)*3+2];
        // priority_queue<pair<float, int> > nn_dist;
        // pair<float, int> max_dist;
        float nn_dist [num_nbrs];
        int nn_idx [num_nbrs];
        int insert_idx = 0;
        int max_idx = 0;
        for (int k=0;k<n;k++){
          if (k==j){
            continue;
          }
          float x2=xyz[(i*n+k)*3+0]-x1;
          float y2=xyz[(i*n+k)*3+1]-y1;
          float z2=xyz[(i*n+k)*3+2]-z1;
          double d=x2*x2+y2*y2+z2*z2;
          if(insert_idx < num_nbrs){   // (LOOP) counter value less than queue capacity, so append
          // if((j < num_nbrs && k <= num_nbrs) || (k < num_nbrs)){  // QUEUE
            // nn_dist.push(make_pair(d,k));
            // max_dist = nn_dist.top();
            nn_dist[insert_idx] = d;
            nn_idx[insert_idx] = k;
            if(d > nn_dist[max_idx]){
              max_idx = insert_idx;
            }
            insert_idx++;
          }
          else if(d < nn_dist[max_idx]){ // LOOP
          // else if(d < max_dist.first){ // QUEUE
            // nn_dist.pop();
            // nn_dist.push(make_pair(d,k));
            // max_dist = nn_dist.top();
            if(d < nn_dist[max_idx]){
              nn_dist[max_idx] = d;
              nn_idx[max_idx] = k;
              // Find new max_idx
              max_idx = 0;
              for(int l=1; l < num_nbrs; l++){
                if(nn_dist[l] > nn_dist[max_idx]){
                  max_idx = l;
                }
              }
            }
          }
        }

        // LOOP - Sort nn_idx by nn_dist
        // Bubble sort nn_idx
        for(int k=0; k<num_nbrs; k++){
          for(int l=num_nbrs-1; l>k; l--){
            if(nn_dist[l] < nn_dist[l-1]){
              // Swap in BOTH nn_idx, nn_dist
              int tmp = nn_idx[l-1];
              nn_idx[l-1] = nn_idx[l];
              nn_idx[l] = tmp;
              float tmpd = nn_dist[l-1];
              nn_dist[l-1] = nn_dist[l];
              nn_dist[l] = tmpd;
            }
          }
        }
        for(int k=0; k<num_nbrs; k++){
          idx[(i*n+j)*num_nbrs+k] = nn_idx[k];
        }
        // for(int k=num_nbrs-1; k>=0; k--){
          // int l = nn_dist.top().second;
          // nn_dist.pop();
          // idx[(i*n+j)*num_nbrs+k] = l;
        // }
        // // Find best-fit plane now
        // vector<Vector3f> points;
        // cout << "Point " << j << "\n";
        // while(!nn_dist.empty()){
        //   int k = nn_dist.top().second;
        //   nn_dist.pop();
        //   Vector3f point;
        //   point(0) = xyz[(i*n+k)*3+0];
        //   point(1) = xyz[(i*n+k)*3+1];
        //   point(2) = xyz[(i*n+k)*3+2];
        //   std::cout << "Nbr: " << k << std::endl;
        //   points.push_back(point);
        // }
        // cout << "\n";
        // pair<Vector3f, Vector3f> plane_pair = best_plane_from_points(points);
        // Vector3f centroid = plane_pair.first;
        // Vector3f plane_normal = plane_pair.second;
        // float dist_from_plane = abs((x1-centroid(0))*plane_normal(0) + (y1-centroid(1))*plane_normal(1) + (z1-centroid(2))*plane_normal(2));
        // dist[i*n+j]=dist_from_plane;
        // // idx[i*n+j]=besti;
      }
    }
}

static void simpleplanedistance(int b, int n, const float *xyz, float *dist, float *offset, float *normal){
  const int num_nbrs = 10;
  for (int i=0; i<b; i+=1){
    for (int j=0; j<n; j+=1){
      float x1=xyz[(i*n+j)*3+0];
      float y1=xyz[(i*n+j)*3+1];
      float z1=xyz[(i*n+j)*3+2];
      // Queue via loop
      float nn_dist [num_nbrs];
      int nn_idx [num_nbrs];
      int insert_idx = 0;
      int max_idx = 0;
      for (int k=0; k < n; k+=1){
        // Compare
        if(j == k)
          continue;
        float x2=xyz[(i*n+k)*3+0]-x1;
        float y2=xyz[(i*n+k)*3+1]-y1;
        float z2=xyz[(i*n+k)*3+2]-z1;
        float d=x2*x2+y2*y2+z2*z2;
        if(insert_idx < num_nbrs){
          nn_dist[insert_idx] = d;
          nn_idx[insert_idx] = k;
          if(d > nn_dist[max_idx]){
            max_idx = insert_idx;
          }
          insert_idx++;
        }
        else if(d < nn_dist[max_idx]){
          nn_dist[max_idx] = d;
          nn_idx[max_idx] = k;
          // Find new max_idx
          max_idx = 0;
          for(int l=1; l < num_nbrs; l++){
            if(nn_dist[l] > nn_dist[max_idx]){
              max_idx = l;
            }
          }
        }
      }

      // Store k nearest nbr indices
      // for(int k=0; k<num_nbrs; k++){
      //   result_i[(i*n+j)*num_nbrs+k] = nn_idx[k];
      // }

      // Init matrices to hold SVD results
      float **a = new float *[num_nbrs];
      for(int k=0; k<num_nbrs; k++)
        a[k] = new float[num_nbrs];
      float **v = new float *[3];
      for(int k=0; k<3; k++)
        v[k] = new float[3];
      float w[3];

      // Copy over nearest nbrs to a
      for(int k=0; k<num_nbrs; k++)
        for(int l=0; l<3; l++)
          a[k][l] = xyz[(i*n+nn_idx[k])*3+l];

      // calculate centroid
      float centroidx = 0;
      float centroidy = 0;
      float centroidz = 0;
      for(int k=0; k<num_nbrs; k++){
        centroidx += a[k][0];
        centroidy += a[k][1];
        centroidz += a[k][2];
      }
      centroidx /= num_nbrs;
      centroidy /= num_nbrs;
      centroidz /= num_nbrs;

      // subtract centroid
      for(int k=0; k<num_nbrs; k++){
        a[k][0] -= centroidx;
        a[k][1] -= centroidy;
        a[k][2] -= centroidz;
      }

      // // Calculate SVD
      // dsvd(a, num_nbrs, 3, w, v);
      // // Find smallest singular value
      // int minidx = (w[0] < w[1]) ? 0 : 1 ;
      // minidx = (w[minidx] < w[2]) ? minidx : 2 ;

      // float nrm[3];
      // nrm[0] = v[0][minidx];
      // nrm[1] = v[1][minidx];
      // nrm[2] = v[2][minidx];
      // normal[(i*n+j)*3+0] = nrm[0];
      // normal[(i*n+j)*3+1] = nrm[1];
      // normal[(i*n+j)*3+2] = nrm[2];

      // // Calculate offset
      // float o = (x1-centroidx)*nrm[0] + (y1-centroidy)*nrm[1] + (z1-centroidz)*nrm[2];
      // offset[i*n+j] = o;
      // dist[i*n+j] = 0.5*o*o;
    }
  }
}

__global__ void PlaneDistanceKernel(int b, int n, const float *xyz, float *dist, float *offset, float *normal){
  const int batch=512;
  // const int num_nbrs = 10;
  __shared__ float buf[batch*3];
  if(blockIdx.x == 0 && blockIdx.y == 0 && threadIdx.x == 0)
    printf("Hi\t");
  for (int i=blockIdx.x;i<b;i+=gridDim.x){
    for (int j=threadIdx.x+blockIdx.y*blockDim.x;j<n;j+=blockDim.x*gridDim.y){
      float x1=xyz[(i*n+j)*3+0];
      float y1=xyz[(i*n+j)*3+1];
      float z1=xyz[(i*n+j)*3+2];
      // Queue via loop
      float nn_dist [NUM_NBRS];
      int nn_idx [NUM_NBRS];
      int insert_idx = 0;
      int max_idx = 0;
      for (int k2=0; k2 < n; k2+=batch){
        // Read into shared buffer
        int end_k=min(n,k2+batch)-k2;
        for (int l=threadIdx.x;l<end_k*3;l+=blockDim.x){
          buf[l]=xyz[(i*n+k2)*3+l];
        }
        __syncthreads();
        // Compare
        for (int k=0;k<end_k;k++){
          if(j == k+k2)
            continue;
          float x2=buf[k*3+0]-x1;
          float y2=buf[k*3+1]-y1;
          float z2=buf[k*3+2]-z1;
          float d=x2*x2+y2*y2+z2*z2;
          if(insert_idx < NUM_NBRS){
            nn_dist[insert_idx] = d;
            nn_idx[insert_idx] = k+k2;
            if(d > nn_dist[max_idx]){
              max_idx = insert_idx;
            }
            insert_idx++;
          }
          else if(d < nn_dist[max_idx]){
            nn_dist[max_idx] = d;
            nn_idx[max_idx] = k+k2;
            // Find new max_idx
            max_idx = 0;
            for(int l=1; l < NUM_NBRS; l++){
              if(nn_dist[l] > nn_dist[max_idx]){
                max_idx = l;
              }
            }
          }
        }
        __syncthreads();
      }

      // Store k nearest nbr indices
      // for(int k=0; k<NUM_NBRS; k++){
      //   result_i[(i*n+j)*NUM_NBRS+k] = nn_idx[k];
      // }

      // Init matrices to hold SVD results
      // float **a = new float *[NUM_NBRS];
      // for(int k=0; k<NUM_NBRS; k++)
      //   a[k] = new float[NUM_NBRS];
      // float **v = new float *[3];
      // for(int k=0; k<3; k++)
      //   v[k] = new float[3];
      float a[NUM_NBRS][NUM_NBRS];
      float v[3][3];
      float w[3];

      // Copy over nearest nbrs to a
      for(int k=0; k<NUM_NBRS; k++)
        for(int l=0; l<3; l++)
          a[k][l] = xyz[(i*n+nn_idx[k])*3+l];

      // calculate centroid
      float centroidx = 0;
      float centroidy = 0;
      float centroidz = 0;
      for(int k=0; k<NUM_NBRS; k++){
        centroidx += a[k][0];
        centroidy += a[k][1];
        centroidz += a[k][2];
      }
      centroidx /= NUM_NBRS;
      centroidy /= NUM_NBRS;
      centroidz /= NUM_NBRS;

      // subtract centroid
      for(int k=0; k<NUM_NBRS; k++){
        a[k][0] -= centroidx;
        a[k][1] -= centroidy;
        a[k][2] -= centroidz;
      }

      // Calculate SVD
      dsvd(a, NUM_NBRS, 3, w, v);
      // Find smallest singular value
      int minidx = (w[0] < w[1]) ? 0 : 1 ;
      minidx = (w[minidx] < w[2]) ? minidx : 2 ;

      float nrm[3];
      nrm[0] = v[0][minidx];
      nrm[1] = v[1][minidx];
      nrm[2] = v[2][minidx];
      normal[(i*n+j)*3+0] = nrm[0];
      normal[(i*n+j)*3+1] = nrm[1];
      normal[(i*n+j)*3+2] = nrm[2];

      // Calculate offset
      float o = (x1-centroidx)*nrm[0] + (y1-centroidy)*nrm[1] + (z1-centroidz)*nrm[2];
      offset[i*n+j] = o;
      dist[i*n+j] = 0.5*o*o;

      // for(int k=0; k<NUM_NBRS; k++)
      //   free((void*) a[k]);
      // free((void**) a);
      // for(int k=0; k<3; k++)
      //   free((void*) v[k]);
      // free((void**) v);
    }
  }
}
void PlaneDistanceKernelLauncher(int b,int n,const float * xyz,float * dist,float * offset, float *normal){
    PlaneDistanceKernel<<<dim3(32,16,1),512>>>(b,n,xyz,dist,offset,normal);
}
__global__ void PlaneDistanceGradKernel(int b,int n,const float *dist,const float *normals,float *grad){
    for (int i=blockIdx.x;i<b;i+=gridDim.x){
      for (int j=threadIdx.x+blockIdx.y*blockDim.x;j<n;j+=blockDim.x*gridDim.y){
        float g=dist[i*n+j];
        float nx=normals[(i*n+j)*3+0];
        float ny=normals[(i*n+j)*3+1];
        float nz=normals[(i*n+j)*3+2];
        grad[(i*n+j)*3+0] = g*nx;
        grad[(i*n+j)*3+1] = g*ny;
        grad[(i*n+j)*3+2] = g*nz;
      }
    }
}
void PlaneDistanceGradKernelLauncher(int b,int n,const float *dist,const float *normals,float *grad){
    hipMemset(grad,0,b*n*3*4);
    PlaneDistanceGradKernel<<<dim3(1,16,1),256>>>(b,n,dist,normals,grad);
}

int main(){
  int b=32;
  int n=4096;
  int m=n;
  float *xyz;
  int *idx;
  hipMallocManaged(&xyz, b*n*3*sizeof(float));
  // Read point cloud
  read_clouds(xyz);

  for(int foo=0; foo<10; foo++){
    printf("%d:\t", foo);
    // usleep(10000);
    float *dist, *offset, *plane, *grad;
    hipError_t err = hipMallocManaged(&dist, b*n*sizeof(float));
    if(dist == nullptr)
      printf("dist null %d\t", err);
    err = hipMallocManaged(&offset, b*n*sizeof(float));
    if(offset == nullptr)
      printf("offset null %d\t", err);
    err = hipMallocManaged(&plane, b*n*3*sizeof(float));
    if(plane == nullptr)
      printf("plane null %d\t", err);
    err = hipMallocManaged(&grad, b*n*3*sizeof(float));
    if(grad == nullptr)
      printf("grad null  %d\t", err);
    printf("Malloc,%f\t", xyz[12287]);
    // usleep(10000);

    // CPU
    // for(int i=0; i<100; i++){
    //   knearestnbr(b, n, xyz, dist, idx);
    // }
    plane[12287] = 0;
    printf("NBefore: %.6f\t", plane[12287]);
    // simpleplanedistance(b, n, xyz, dist, offset, plane);
    // GPU
    PlaneDistanceKernelLauncher(b, n, xyz, dist, offset, plane);
    hipDeviceSynchronize();
    printf("NAfter: %.6f\t", plane[12287]);
    // printf("Normal: %.6f\t", plane[0]);
    // usleep(10000);
    PlaneDistanceGradKernelLauncher(b, n, offset, plane, grad);
    hipDeviceSynchronize();
    printf("Grad: %.6f\t", grad[0]);
    // usleep(10000);

    hipFree(dist);
    hipFree(offset);
    hipFree(plane);
    hipFree(grad);
    printf("Free\n");
    // usleep(10000);
  }

  // FILE *pfile;
  // // pfile = fopen("ans_knearestnbr_gpu.txt","w");
  // // for (int i=0; i < b*n; i+=1){
  // //   for (int k=0; k < num_nbrs; k++){
  // //     fprintf(pfile, "%d ", idx[i*num_nbrs + k]+1);
  // //   }
  // //   fprintf(pfile, "\n");
  // // }
  // // fclose(pfile);

  // pfile = fopen("ans_planedist_gpu.txt","w");
  // for (int i=0; i < b*n; i+=1){
  //   fprintf(pfile, "%f\n", dist[i]);
  // }
  // fclose(pfile);

  // pfile = fopen("ans_planenormals_gpu.txt","w");
  // for (int i=0; i < b*n; i+=1){
  //   fprintf(pfile, "%.3f %.3f %.3f\n", plane[i*3+0], plane[i*3+1], plane[i*3+2]);
  // }
  // fclose(pfile);

  // pfile = fopen("ans_grad_gpu.txt","w");
  // for (int i=0; i < b*n; i+=1){
  //   fprintf(pfile, "%.5f %.5f %.5f\n", grad[i*3+0], grad[i*3+1], grad[i*3+2]);
  // }
  // fclose(pfile);

  hipFree(xyz);

  return 0;
}
